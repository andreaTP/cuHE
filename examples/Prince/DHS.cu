#include "hip/hip_runtime.h"
/* 
 *	The MIT License (MIT)
 *	Copyright (c) 2013-2015 Wei Dai
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy
 *	of this software and associated documentation files (the "Software"), to deal
 *	in the Software without restriction, including without limitation the rights
 *	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *	copies of the Software, and to permit persons to whom the Software is
 *	furnished to do so, subject to the following conditions:
 *
 *	The above copyright notice and this permission notice shall be included in
 *	all copies or substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 *	THE SOFTWARE.
 */

#include "DHS.h"
#include "../../cuhe/CuHE.h"
using namespace cuHE;

///////////////////////////////////////////////////////////////////////////////
// @class CuDHS
///////////////////////////////////////////////////////////////////////////////
//// Constructor //////////////////////////////////////////
CuDHS::CuDHS(int d, int p, int w, int min, int cut, int m) {
	setParameters(d, p, w, min, cut, m);
	coeffMod_ = new ZZ[param.depth];
	pk_ = new ZZX[param.depth];
	sk_ = new ZZX[param.depth];
	if (param.logRelin > 0) {
		ek_ = new ZZX[param.numEvalKey];
	}
	else {
		ek_ = NULL;
	}
	genPolyMod_(); // generate polynomial modulus
	initCuHE(coeffMod_, polyMod_); // create polynomial ring
	B = 1;
	keyGen(); // key generation
	numSlot_ = param.modLen/factorDegree();
	batcher = new Batcher(polyMod_, param.modLen/numSlot_, numSlot_); // setup batching
}
CuDHS::~CuDHS() {
	clear(polyMod_);
	delete [] coeffMod_;
	delete [] pk_;
	delete [] sk_;
	if (ek_ != NULL)
		delete [] ek_;
//	delete batcher;
}
ZZX CuDHS::polyMod() { return polyMod_;};
ZZ* CuDHS::coeffMod() { return coeffMod_;};
int CuDHS::numSlot() { return numSlot_;};
ZZX* CuDHS::ek() { return ek_;};
//// Primitives ///////////////////////////////////////////
void CuDHS::keyGen() {
	genPkSk();
	if (param.logRelin > 0) {
		genEk();
	}
}
void CuDHS::encrypt(ZZX& out, ZZX in, int lvl) {
	ZZX s, e, t;
	s = sample();
	e = sample();
	coeffReduce(s, s, lvl);

	mulZZX(t, pk_[lvl], s, lvl, 0, 0);
//	t = pk_[lvl]*s;
//	t %= polyMod_;
//	coeffReduce(t, t, lvl);

	t += e*param.modMsg+in;
	coeffReduce(t, t, lvl);
	out = t;
}
void CuDHS::decrypt(ZZX& out, ZZX in, int lvl, int maxMulPath) {
	ZZ x;
	ZZX t = in;
	coeffReduce(t, t, lvl);
	if (param.logRelin > 0)
		for (int i=0; i<maxMulPath; i++)
			mulZZX(t, t, sk_[lvl], lvl, 0, 0);
	else
		mulZZX(t, t, sk_[lvl], lvl, 0, 0);
//	ZZX t = sk_[lvl]*in;
//	t %= polyMod_;
//	coeffReduce(t, t, coeffMod_[lvl]);

	clear(out);
	for (int i=0; i<=deg(t); i++) {
		x = coeff(t, i);
		if (x > ((coeffMod_[lvl]-1)/2))
			x -= coeffMod_[lvl];
		SetCoeff(out, i, (x%param.modMsg));
	}
}
void CuDHS::unbalance(ZZX& x, int lvl) {
	ZZ tp, q = coeffMod_[lvl];
	for (int i=0; i<=deg(x); i++) {
		tp = coeff(x, i);
		if (tp < 0)
			tp += q;
		SetCoeff(x, i, tp);
	}
}
void CuDHS::balance(ZZX& x, int lvl) {
	ZZ tp, q = coeffMod_[lvl];
	for (int i=0; i<=deg(x); i++) {
		tp = coeff(x, i);
		if (tp > ((q-1)/2))
			tp -= q;
		SetCoeff(x, i, tp);
	}
}

//// Tools ////////////////////////////////////////////////
int CuDHS::factorDegree() {
	int ret = 1;
	while ( (power(to_ZZ(param.modMsg), ret)-1)%param.mSize != 0 )
		ret++;
	cout<<ret<<endl;
	return ret;
}

void CuDHS::genPolyMod_() {
	int s;
	polyMod_ = 1;
	ZZX *t_vec = new ZZX[param.mSize];
	int *s_vec = new int[param.mSize];
	for (int i=0; i<param.mSize; i++)
		s_vec[i] = 0;
	for (int d=1; d<=param.mSize; d++) {
		if (GCD(d, param.mSize) == d) {
			ZZX t;
			SetCoeff(t, 0 , -1);
			SetCoeff(t, param.mSize/d, 1);
			s = mobuisFunction(d);
			t_vec[d-1] = t;
			s_vec[d-1] = s;
		}
	}
	for (int i=0; i<param.mSize; i++)
		if (s_vec[i] == 1)
			polyMod_ *= t_vec[i];
	for (int i=0; i<param.mSize; i++)
		if (s_vec[i] == -1)
			polyMod_ /=  t_vec[i];
	delete [] t_vec;
	delete [] s_vec;
}
void CuDHS::genPkSk() {
	// sample
	ZZX f, g, ft, f_inv;
	coeffReduce(polyMod_, polyMod_, 0);//
	bool isfound = false;
	while (!isfound) {
		isfound = true;
		ft = sample();
		f = ft*param.modMsg + 1;
		coeffReduce(f, f, 0);//
		findInverse(f_inv, f, coeffMod_[0], isfound);
		coeffReduce(f_inv, f_inv, 0);
	}
	isfound = false;
	g = sample();
	coeffReduce(g, g, 0);
	// sk[0], pk[0] from (f, g, f_inv)
	sk_[0] = f;
	mulZZX(pk_[0], g, f_inv, 0, 0, 0); // pk[0] = g*f_inv, reduce
//	cout<<pk_[0]<<endl;
//	pk_[0] = g*f_inv;
//	pk_[0] %= polyMod_;
//	coeffReduce(pk_[0], pk_[0], 0);

	pk_[0] *= param.modMsg;
	coeffReduce(pk_[0], pk_[0], 0);
	coeffReduce(sk_[0], sk_[0], 0);
	for(int i=1; i<param.depth; i++){
		sk_[i] = sk_[i-1];
		coeffReduce(sk_[i], sk_[i], i);
		pk_[i] = pk_[i-1];
		coeffReduce(pk_[i], pk_[i], i);
	}
}
void CuDHS::genEk() {
	ZZX tk = sk_[0];
	ZZ tw =to_ZZ(1);
	ZZ w = to_ZZ(1)<<param.logRelin;
	ZZX s, e, result, tp;
	for (int i=0; i<param.numEvalKey; i++) {
		tp = tk*tw;
		s = sample();
		e = sample();
		coeffReduce(s, s, 0);
		coeffReduce(e, e, 0);
		coeffReduce(tp, tp, 0);
		mulZZX(ek_[i], pk_[0], s, 0, 0, 0);
		ek_[i] += e*param.modMsg+tp;
//		ek2.key[i] = pk[0]*s + e*p + tp;
//		Arith_PolyReduce(ek2.key[i], ek2.key[i]);

		coeffReduce(ek_[i], ek_[i], 0);
		tw *= w;
	}
	initRelinearization(ek_);
}
void CuDHS::coeffReduce(ZZX& out, ZZX in, int lvl) {
	coeffReduce(out, in, coeffMod_[lvl]);
}
void CuDHS::coeffReduce(ZZX& out, ZZX in, ZZ q) {
	clear(out);
	for (int i=0; i<=deg(in); i++)
		SetCoeff(out, i, coeff(in,i)%q);
}
ZZX CuDHS::sample(){
	ZZX ret;
	for (int i=0; i<param.modLen; i++)
		SetCoeff(ret, i, RandomBnd(to_ZZ(2*B+1))-B);
	return ret;
}
void CuDHS::findInverse(ZZX &f_inv, ZZX &f, ZZ &q, bool &isfound) {
	ZZ_p::init(q);
	ZZ_pX phi;
	phi = to_ZZ_pX(polyMod_);
	ZZ_pE::init(phi);

	ZZ_pE f_, f_inv_;
	f_ = to_ZZ_pE(to_ZZ_pX(f));
	try{ f_inv_ = inv(f_); }
	catch(runtime_error &e)
	{
		isfound = false;
	}
	ZZ_pX tp = rep(f_inv_);
	for(int i=0; i<param.modLen; i++)
		SetCoeff(f_inv, i, rep(coeff(tp, i)));
}
int CuDHS::mobuisFunction(int n) {
	int t, primes;
	primes = 0;

	if (n == 1)
		return 1;
	else {
		for (int i=2; i<=n; i++) {
			if (ProbPrime(i)) {
				if (GCD(i,n) == i) {
					t=n/i;
					primes++;
					if (GCD(i, t) == i)
						return 0;
				}
			}
		}
		if (primes%2 == 0)
			return 1;
		else
			return -1;
	}
}

///////////////////////////////////////////////////////////////////////////////
// @class Batcher
///////////////////////////////////////////////////////////////////////////////
//// Constructor //////////////////////////////////////////
Batcher::Batcher(ZZX polymod, int f_degree, int f_size) {
	if (param.modMsg != 2) {
		cout<<"Error: This Batcher code only supports 1-bit messages."<<endl;
		terminate();
	}
	ZZ_p::init(to_ZZ(2));
	SetModulus(polymod);
	ComputeFactors(f_degree, f_size);
	CalculateMs();
	CalculateNs();
	CalculateMxNs();
}
Batcher::~Batcher() {}

void Batcher::SetModulus(ZZX m) {
	modulus = to_ZZ_pX(m);
}
void Batcher::ComputeFactors(int f_degree, int f_size) {
	factors.SetLength(f_size);
	int s = 1<<f_degree;
	ZZ_pX *list = new ZZ_pX[s];
	for (int i=0; i<s; i++)
		list[i] = to_ZZ_pX(num2ZZX(i+s));

	int j=0;
	ZZ_pX t1 = modulus;
	ZZ_pX comp, remin, quo;
	SetCoeff(comp, 0, 0);
	for (int i=0; i<s; i++) {
		DivRem(quo, remin, t1, list[i]);
		if (remin == comp) {
			t1 = quo;
			factors[j] = list[i];
			j++;
		}
	}
	size = factors.length();
}
void Batcher::CalculateMs() {
	ZZ_pX temp;
	M.SetLength(size);

	for (int i=0; i<size; i++) {
		M[i] = modulus;
		M[i] = M[i] / factors[i];
	}
}
void Batcher::CalculateNs() {
	ZZ_pX mi;
	N.SetLength(size);
	for (int i=0; i<size; i++) {
		mi = factors[i];
		ZZ_pE::init(mi);

		ZZ_pE t = to_ZZ_pE((M[i])%mi);
		ZZ_pE ti = inv(t);

		N[i] = rep(ti);
	}
}
void Batcher::CalculateMxNs() {
	MxN.SetLength(size);
	for (int i=0; i<size; i++)
		MxN[i] = (M[i]*N[i])%modulus;
}
void Batcher::encode(ZZX &poly, ZZX mess) {
	ZZ_p::init(to_ZZ(2));
	ZZ_pX res;
	SetCoeff(res, 0, 0);
	for (int i=0; i<size; i++)
		if (coeff(mess,i) == 1)
			res = res + MxN[i];
	res %= modulus;
	poly = to_ZZX(res);
}
void Batcher::decode(ZZX &mess, ZZX poly) {
	ZZ t;
	ZZ_pX mess_p, tm;
	mess_p = to_ZZ_pX(poly);
	clear(mess);
	for (int i=0; i<size; i++) {
		tm = mess_p%factors[i];
		t = rep(coeff(tm, 0));
		t %= to_ZZ(2);
		SetCoeff(mess, i, t);
	}
}

ZZX Batcher::num2ZZX(int num){
	ZZX res;
	SetCoeff(res, 0 , 0);
	if(num == 0)
		return res;
	else{
		for(int i=0; i<32; i++)
			SetCoeff(res, i, (num>>i)%2);
	}
	return res;
}
